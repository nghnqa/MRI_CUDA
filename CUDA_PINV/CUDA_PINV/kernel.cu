#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <hipsolver.h>
#include <hipblas.h>
/////////////////////
using namespace std;
#define M 3000
#define N 3000
/////////////////////
int readInput(float *A);
/////////////////////
int main()
{		 
	float *h_A, *h_U, *h_S, *h_Vt, *d_A, *d_U, *d_S, *d_Vt, *work;
	int *devInfo, work_size = 0, devInfo_h = 0;

	hipsolverHandle_t solver_handle; 
	hipsolverDnCreate(&solver_handle);
	hipsolverDnSgesvd_bufferSize(solver_handle, M, N, &work_size);

	h_A = (float*)malloc(M*N*sizeof(float));
	h_U = (float*)malloc(M*M*sizeof(float));
	h_Vt = (float*)malloc(N*N*sizeof(float));
	h_S = (float*)malloc(N*sizeof(float));

	hipMalloc((void**)&d_A,M*N*sizeof(float));
	hipMalloc((void**)&d_U,M*M*sizeof(float));
	hipMalloc((void**)&d_Vt,N*N*sizeof(float));
	hipMalloc((void**)&d_S,N*sizeof(float));
	hipMalloc((void**)&devInfo, sizeof(int));
	hipMalloc(&work, work_size * sizeof(float));

	readInput(h_A);
	hipMemcpy(d_A,h_A,M*N*sizeof(float), hipMemcpyHostToDevice);
	hipsolverDnSgesvd(solver_handle, 'A', 'A', M, N, d_A, M, d_S, d_U, M, d_Vt, N, work, work_size, NULL, devInfo);
	hipDeviceSynchronize();
	hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_U, d_U, M*M*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_Vt, d_Vt,N*N*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_S, d_S,N*sizeof(float), hipMemcpyDeviceToHost);
	std::cout << "devInfo = " << devInfo_h << "\n";
	std::cout << "SVD successful\n\n";

	hipFree(d_A); hipFree(d_U); hipFree(d_Vt); hipFree(d_S); hipFree(devInfo); hipFree(work);
	hipsolverDnDestroy(solver_handle);
    
	///////////////////////////////////////////////////

	hipblasHandle_t cublas_handler;
	hipblasCreate(&cublas_handler);
	hipblasDestroy(cublas_handler);
	
	return 0;
}
/////////////////////
int readInput(float *A)
{
	fstream input_matrix;
	input_matrix.open("E:/MRI/test_matrix.txt", ios::in);
	if (input_matrix.is_open())
	{
		for (unsigned int i = 0; i < M*N; i++)
		{
			input_matrix >> A[i];
		}
		input_matrix.close();
		return 0;
	}
	else
	{
		cout << "readInput => ERROR in opening file!" << endl;
		return -1;
	}
}