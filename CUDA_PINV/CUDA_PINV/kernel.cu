#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <hipsolver.h>
#include <hipblas.h>
/////////////////////
using namespace std;
#define M 3000
#define N 3000
/////////////////////
int readInput(float *A);
/////////////////////
int main()
{		 
	float *h_A, *h_U, *h_S, *h_S2, *h_Vt, *d_A, *d_U, *d_S, *d_Vt, *work;
	int *devInfo, work_size = 0, devInfo_h = 0;
	
	hipsolverHandle_t solver_handle; 
	hipsolverDnCreate(&solver_handle);
	hipsolverDnSgesvd_bufferSize(solver_handle, M, N, &work_size);

	h_A = (float*)malloc(M*N*sizeof(float));
	h_U = (float*)malloc(M*M*sizeof(float));
	h_Vt = (float*)malloc(N*N*sizeof(float));
	h_S = (float*)malloc(N*sizeof(float)); // this is used for cusolverDn 
	h_S2 = (float*)calloc(M*N,sizeof(float)); // this is full matrixy version of S => we need it to be 0 at first

	hipMalloc((void**)&d_A,M*N*sizeof(float));
	hipMalloc((void**)&d_U,M*M*sizeof(float));
	hipMalloc((void**)&d_Vt,N*N*sizeof(float));
	hipMalloc((void**)&d_S,N*sizeof(float));
	hipMalloc((void**)&devInfo, sizeof(int));
	hipMalloc(&work, work_size * sizeof(float));

	readInput(h_A);
	hipMemcpy(d_A,h_A,M*N*sizeof(float), hipMemcpyHostToDevice);
	hipsolverDnSgesvd(solver_handle, 'A', 'A', M, N, d_A, M, d_S, d_U, M, d_Vt, N, work, work_size, NULL, devInfo);
	hipDeviceSynchronize();
	hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_U, d_U, M*M*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_Vt, d_Vt,N*N*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_S, d_S,N*sizeof(float), hipMemcpyDeviceToHost);
	std::cout << "devInfo = " << devInfo_h << "\n";
	std::cout << "SVD successful\n\n";

	hipFree(d_A); hipFree(d_S); hipFree(devInfo); hipFree(work);
	hipsolverDnDestroy(solver_handle);
    
	/////////////Repairing S for multiplication///////////////
	float *d_S2;
	hipMalloc((void**)&d_S2, M*N*sizeof(float));
	for (int i = 0; i < N; i++)
	{
		if (h_S[i] != 0)
		{
			h_S2[i*(N + 1)] = 1.0 / h_S[i];
		}
	}
	hipMemcpy(d_S2, h_S2, N*M*sizeof(float), hipMemcpyHostToDevice);
	free(h_S); free(h_S2);
	/////////////cublas_V2 for matrix-matrix multiplication//////////
	hipblasHandle_t cublas_handler;
	float *d_KQ1, *d_KQ2, *alpha, *beta;
	alpha = (float*)malloc(sizeof(float)); *alpha = 1.0f;
	beta = (float*)malloc(sizeof(float)); *beta = 1.0f;
	hipblasCreate(&cublas_handler);
	hipMalloc((void**)&d_KQ1, N*M*sizeof(float));					 
	hipMemset(d_KQ1, 0, N*M*sizeof(float));
	hipblasSgemm(cublas_handler, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, N, alpha , d_Vt, N, d_S2, M, beta, d_KQ1, N);
	hipMalloc((void**)&d_KQ2, N*M*sizeof(float));
	hipMemset(d_KQ1, 0, N*M*sizeof(float));												  
	hipblasSgemm(cublas_handler, HIPBLAS_OP_T, HIPBLAS_OP_T, N, M, M, alpha, d_KQ1, N, d_U, M, beta, d_KQ2, N);
	hipFree(d_KQ1); hipFree(d_Vt); hipFree(d_S2); 
	free(alpha); free(beta);
	hipblasDestroy(cublas_handler);
	return 0;
}
/////////////////////
int readInput(float *A)
{
	fstream input_matrix;
	input_matrix.open("E:/MRI/test_matrix.txt", ios::in);
	if (input_matrix.is_open())
	{
		for (unsigned int i = 0; i < M*N; i++)
		{
			input_matrix >> A[i];
		}
		input_matrix.close();
		return 0;
	}
	else
	{
		cout << "readInput => ERROR in opening file!" << endl;
		return -1;
	}
}